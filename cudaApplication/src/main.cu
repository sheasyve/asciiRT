#include "hip/hip_runtime.h"
#include "utils/main_util.cuh"

// Scene settings
int w = 180, h = 180;

// Camera settings
const float focal_length = 2.16;
const float field_of_view = 0.7854; // 45 degrees
const Eigen::Vector3f camera_position(0, 0, -100);

// Rotation settings
bool rotate = true;

// Lights
std::vector<Eigen::Vector3f> light_positions;
std::vector<Eigen::Vector4f> light_colors;

// Meshes
std::vector<Mesh> meshes;

void gen_rays(int w, int h, std::vector<Eigen::Vector3f> &ray_origins, std::vector<Eigen::Vector3f> &ray_directions)
{
    const float aspect_ratio = float(w) / float(h);
    const float y = (((focal_length)*sin(field_of_view / 2)) / sin((180 - (90 + ((field_of_view * (180 / M_PI) / 2)))) * (M_PI / 180)));
    const float x = (y * aspect_ratio);
    Eigen::Vector3f image_origin(-x, y, camera_position[2] - focal_length);
    Eigen::Vector3f x_displacement(2.0 / w * x, 0, 0);
    Eigen::Vector3f y_displacement(0, -2.0 / h * y, 0);
    for (int j = 0; j < h; j++)
    {
        for (int i = 0; i < w; i++)
        {
            Eigen::Vector3f pixel_center = image_origin + (i + 0.5) * x_displacement + (j + 0.5) * y_displacement;
            ray_origins.push_back(camera_position);
            ray_directions.push_back((camera_position - pixel_center).normalized());
        }
    }
}

void setup_scene(int argc, char *argv[])
{
    load_meshes(argc, argv, meshes);
    float rX = -.05, rY = .8, rZ = .05; // Rotation IN RADIANS
    if (meshes.size() > 0 && rotate)
        meshes[0].triangles = rotate_mesh(meshes[0], rX, rY, rZ); // Rotate mesh 1
    // meshes[0].triangles = translate_mesh(meshes[0],5,5,5);//Translate mesh 1
    light_colors.emplace_back(0.8, 0.8, 0.8, 1); // Light 1
    light_positions.emplace_back(0, 5, -30);
    light_colors.emplace_back(0.4, 0.4, 0.4, 1); // Light 2
    light_positions.emplace_back(10, -5, -20);
    light_colors.emplace_back(0.3, 0.3, 0.3, 1); // Light 3
    light_positions.emplace_back(10, 5, 20);
    light_colors.emplace_back(0.2, 0.2, 0.2, 1); // Light 4
    light_positions.emplace_back(-10, 20, -30);
}

int main(int argc, char *argv[])
{
    auto start = std::chrono::high_resolution_clock::now();
    std::vector<Eigen::Vector3f> ray_origins, ray_directions;
    gen_rays(w, h, ray_origins, ray_directions);
    setup_scene(argc, argv);
    float *output = h_raytrace(ray_origins, ray_directions, meshes, w, h, light_positions, light_colors);
    print_scene_in_ascii(output, w, h);
    std::cout << "Runtime: " << std::chrono::duration<float>(std::chrono::high_resolution_clock::now() - start).count() << " seconds" << std::endl;
    return 0;
}
