#include "hip/hip_runtime.h"
#include "ascii_print.cuh"

void print_scene_in_ascii(float* color, int w, int h) {
    // ASCII characters for brightness levels
    const std::string brightness_chars = " `.-':_,^=;><+!rc*/z?sLTv)J7(|Fi{C}fI31tlu[neoZ5Yxjya]2ESwqkP6h9d4VpOGbUAKXHm8RD#$Bg0MNWQ%&@";
    const int l = brightness_chars.size() - 1;
    // Print the model in ASCII
    for (int j = h; j >= 0; --j) {
        for (int i = 0; i < w; ++i) {
            double brightness = color[j * w + i];
            brightness = std::max(0.0, std::min(1.0, brightness)); // Clamp brightness between 0 and 1
            char c = brightness_chars[static_cast<int>(l * brightness)];
            std::cout << c;
        }
        std::cout << "\n";
    }
}
