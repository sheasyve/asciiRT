#include "hip/hip_runtime.h"
#include "cuda_rt.cuh"

__global__ void d_raytrace(
    Eigen::Vector3f* ray_origins, Eigen::Vector3f* ray_directions, 
    BvhTree::Node* nodes, int root_index, Triangle* triangles,
    float* output,
    int width, int height,
    Eigen::Vector3f* light_positions,
    Eigen::Vector4f* light_colors, int num_lights
) {
    //Ray init
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;
    int idx = y * width + x;
    Eigen::Vector3f origin = ray_origins[idx];
    Eigen::Vector3f direction = ray_directions[idx];
    // Phong shading parameters
    float diffuse_intensity = 0.4;
    float specular_intensity = 0.4;
    float reflection_coefficient = 0.5; 
    float shine = 32.0;
    float a = 1.0, b = 0.1, c = 0.01;
    float brightness = 0.0;
    int max_depth = 3; //Max reflections
    for (int depth = 0; depth < max_depth; depth++) {//Perform RT up to depth times for each reflection
        float local_brightness = 0.005, min_t = INF;
        int mindex = find_closest_triangle(origin, direction, nodes, root_index, triangles, min_t);
        if (mindex == -1) {brightness += 0.0;break;}
        // Compute intersection point and normal
        Eigen::Vector3f p = origin + direction * min_t;
        Triangle closest = triangles[mindex];
        Eigen::Vector3f N = closest.normal();
        N.normalize();
        Eigen::Vector3f V = -direction;
        V.normalize();
        for (int i = 0; i < num_lights; i++) {//For each light
            Eigen::Vector3f L = light_positions[i] - p;
            float d = L.norm();
            L.normalize();
            //Shadow ray cast
            Eigen::Vector3f shadow_ray_origin = p + N * 1e-4;
            float shadow_ray_t = d; // Maximum distance to check (distance to light)
            int shadow_mindex = find_closest_triangle(shadow_ray_origin, L, nodes, root_index, triangles, shadow_ray_t);
            bool in_shadow = (shadow_mindex != -1 && shadow_ray_t > 0.0);
            float attenuation = in_shadow ? 0.0 : (1.0 / (a + b * d + c * d * d));
            Eigen::Vector3f light_rgb = light_colors[i].head<3>();
            // Diffuse
            float lambertian = fmax(N.dot(L), 0.0);
            local_brightness += attenuation * diffuse_intensity * lambertian * light_rgb.norm();
            // Specular
            Eigen::Vector3f R = (2.0 * N.dot(L) * N - L).normalized();
            float spec_angle = fmax(R.dot(V), 0.0);
            float specular = pow(spec_angle, shine);
            local_brightness += attenuation * specular_intensity * specular * light_rgb.norm();
        }
        local_brightness = fmin(local_brightness, 1.0);
        brightness += pow(reflection_coefficient, depth) * local_brightness;
        direction = direction - 2.0 * direction.dot(N) * N;//Calculate new ray direction from reflection
        direction.normalize();
        origin = p + direction * 1e-4;
    }
    output[idx] = brightness;
}

float* h_raytrace(
    std::vector<Eigen::Vector3f> ray_origins,
    std::vector<Eigen::Vector3f> ray_directions, 
    std::vector<Mesh> meshes,
    int width, int height,
    std::vector<Eigen::Vector3f> light_positions,
    std::vector<Eigen::Vector4f> light_colors
) {
    int size = width * height;
    int num_lights = static_cast<int>(light_positions.size());
    std::vector<Triangle> triangles = get_triangles(meshes);
    int num_triangles = static_cast<int>(triangles.size());
    // Build BVH
    BvhTree bvh(triangles);
    std::vector<BvhTree::Node>& nodes = bvh.nodes;
    int tree_size = static_cast<int>(nodes.size());
    int root = bvh.root;
    float* h_output = new float[size];
    //Device pointers
    Eigen::Vector3f* d_ray_origins = nullptr;
    Eigen::Vector3f* d_ray_directions = nullptr;
    Triangle* d_triangles = nullptr;
    BvhTree::Node* d_nodes = nullptr;
    float* d_output = nullptr;
    Eigen::Vector3f* d_lights = nullptr;
    Eigen::Vector4f* d_light_colors = nullptr;
    //Allocate
    hipMalloc((void**)&d_ray_origins, size * sizeof(Eigen::Vector3f));
    hipMalloc((void**)&d_ray_directions, size * sizeof(Eigen::Vector3f));
    hipMalloc((void**)&d_triangles, num_triangles * sizeof(Triangle));
    hipMalloc((void**)&d_nodes, tree_size * sizeof(BvhTree::Node));
    hipMalloc((void**)&d_output, size * sizeof(float));
    hipMalloc((void**)&d_lights, num_lights * sizeof(Eigen::Vector3f));
    hipMalloc((void**)&d_light_colors, num_lights * sizeof(Eigen::Vector4f));
    //Copy
    hipMemcpy(d_ray_origins, ray_origins.data(), size * sizeof(Eigen::Vector3f), hipMemcpyHostToDevice);
    hipMemcpy(d_ray_directions, ray_directions.data(), size * sizeof(Eigen::Vector3f), hipMemcpyHostToDevice);
    hipMemcpy(d_triangles, triangles.data(), num_triangles * sizeof(Triangle), hipMemcpyHostToDevice);
    hipMemcpy(d_nodes, nodes.data(), tree_size * sizeof(BvhTree::Node), hipMemcpyHostToDevice);
    hipMemcpy(d_lights, light_positions.data(), num_lights * sizeof(Eigen::Vector3f), hipMemcpyHostToDevice);
    hipMemcpy(d_light_colors, light_colors.data(), num_lights * sizeof(Eigen::Vector4f), hipMemcpyHostToDevice);
    //Kernel
    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);
    d_raytrace<<<gridDim, blockDim>>>(
        d_ray_origins, d_ray_directions, d_nodes, root, d_triangles,
        d_output,
        width, height,
        d_lights, d_light_colors, num_lights
    );
    //Copy back and free
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("CUDA error. %s\n", hipGetErrorString(err));
    hipMemcpy(h_output, d_output, size * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_ray_origins);
    hipFree(d_ray_directions);
    hipFree(d_triangles);
    hipFree(d_nodes);
    hipFree(d_output);
    hipFree(d_lights);
    hipFree(d_light_colors);
    return h_output;
}

